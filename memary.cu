#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstring>
#include <iostream>
using namespace std;

__device__ int ga = 8;
__device__ int gb;

__global__ void var() {
    int a = 0;          // 寄存器
    int s[1000] = {0};  // 本地内存
    printf("%p %p", &a, &s);
}

int main(int argc, char** argv) {
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, dev);
    cout << prop.name << endl;
    cout << "num of mutipro:" << prop.multiProcessorCount << endl;
    cout << "max num threads of per block:" << prop.maxThreadsPerBlock << endl;
    cout << "max num threads of per mutipro:" << prop.maxThreadsPerMultiProcessor << endl;
    cout << "max num warps of per mutipro:" << prop.maxThreadsPerMultiProcessor / 32 << endl;
    int test = 100;
    hipMemcpyToSymbol(HIP_SYMBOL(gb), &test, sizeof(int));
    var<<<1, 8>>>();
}