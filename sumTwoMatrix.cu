#include <hip/hip_runtime.h>

#include <cstring>
#include <iostream>
using namespace std;

void initval(int s[], int n) {
    for (int i = 0; i < n; i++) {
        s[i] = 1;
    }
}

__global__ void sumTwoArrayOnGpu(int* ga, int* gb, int* gres) {
    int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    gres[i] = ga[i] + gb[i];
}

void sumTwoMatrix(int tx, int ty) {
    int l = 1024;
    int n = l * l;
    int nbyte = sizeof(int) * n;

    int* a = new int[n];
    int* b = new int[n];
    int* res = new int[n];
    initval(a, n);
    initval(b, n);

    int *ga, *gb, *gres;
    hipMalloc(&ga, nbyte);
    hipMalloc(&gb, nbyte);
    hipMalloc(&gres, nbyte);

    hipMemcpy(ga, a, nbyte, hipMemcpyHostToDevice);
    hipMemcpy(gb, b, nbyte, hipMemcpyHostToDevice);
    hipMemcpy(gres, a, nbyte, hipMemcpyHostToDevice);

    dim3 block(tx, ty);
    dim3 grid(l / tx, l / ty);
    cout << tx << " " << ty << " " << l / tx << " " << l / ty;
    sumTwoArrayOnGpu<<<grid, block>>>(ga, gb, gres);
    hipMemcpy(res, gres, nbyte, hipMemcpyDeviceToHost);

    cout << res[n - l] << endl;

    hipFree(ga);
    hipFree(gb);
    hipFree(gres);
    delete[] a;
    delete[] b;
    delete[] res;
}

int main(int argc, char** argv) {
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, dev);
    cout << prop.name << endl;
    cout << "num of mutipro:" << prop.multiProcessorCount << endl;
    cout << "max num threads of per block:" << prop.maxThreadsPerBlock << endl;
    cout << "max num threads of per mutipro:" << prop.maxThreadsPerMultiProcessor << endl;
    cout << "max num warps of per mutipro:" << prop.maxThreadsPerMultiProcessor / 32 << endl;

    int tx = atoi(argv[1]);
    int ty = atoi(argv[2]);
    sumTwoMatrix(tx, ty);
}