#include <hip/hip_runtime.h>

#include <cstring>
#include <iostream>
using namespace std;

void initval(int s[], int n) {
    for (int i = 0; i < n; i++) {
        s[i] = 1;
    }
}

__global__ void sumArrayOnGpu(int* ga, int* gb, int* gres, int tx, int ty, int bx, int by) {
    int i = (blockIdx.y * by + blockIdx.x) * tx * ty + threadIdx.y * ty + threadIdx.x;
    gres[i] = ga[i] + gb[i];
}

void test(int tx, int ty) {
    int l = 32 * 1024;
    int n = l * l;
    int nbyte = sizeof(int) * n;

    int* a = new int[n];
    int* b = new int[n];
    int* res = new int[n];
    initval(a, n);
    initval(b, n);

    int *ga, *gb, *gres;
    hipMalloc(&ga, nbyte);
    hipMalloc(&gb, nbyte);
    hipMalloc(&gres, nbyte);

    hipMemcpy(ga, a, nbyte, hipMemcpyHostToDevice);
    hipMemcpy(gb, b, nbyte, hipMemcpyHostToDevice);

    dim3 block(tx, ty);
    dim3 grid(l / tx, l / ty);

    sumArrayOnGpu<<<grid, block>>>(ga, gb, gres, tx, ty, l / tx, l / ty);
    hipMemcpy(res, gres, nbyte, hipMemcpyDeviceToHost);

    cout << res[10666] << endl;

    hipFree(ga);
    hipFree(gb);
    hipFree(gres);
    delete[] a;
    delete[] b;
    delete[] res;
}

int main(int argc, char** argv) {
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, dev);
    cout << prop.name << endl;
    cout << "num of mutipro:" << prop.multiProcessorCount << endl;
    cout << "max num threads of per block:" << prop.maxThreadsPerBlock << endl;
    cout << "max num threads of per mutipro:" << prop.maxThreadsPerMultiProcessor << endl;
    cout << "max num warps of per mutipro:" << prop.maxThreadsPerMultiProcessor / 32 << endl;

    int tx = atoi(argv[1]);
    int ty = atoi(argv[2]);
    cout << tx << " " << ty;
    test(tx, ty);
}