#include <hip/hip_runtime.h>

#include <iostream>
using namespace std;

void initval(int s[], int n) {
    for (int i = 0; i < n; i++) {
        s[i] = i;
    }
}

__global__ void sumArrayOnGpu(int* ga, int* gb, int* gres) {
    int i = threadIdx.x;
    gres[i] = ga[i] + gb[i];
}

int main() {
    int dev = 0;
    // cout << cudaGetDeviceCount();
    hipSetDevice(dev);
    int n = 32;
    int nbyte = sizeof(int) * n;

    int* a = new int[n];
    int* b = new int[n];
    int* res = new int[n];
    initval(a, n);
    initval(b, n);

    int *ga, *gb, *gres;
    hipMalloc(&ga, nbyte);
    hipMalloc(&gb, nbyte);
    hipMalloc(&gres, nbyte);

    hipMemcpy(ga, a, nbyte, hipMemcpyHostToDevice);
    hipMemcpy(gb, b, nbyte, hipMemcpyHostToDevice);

    dim3 block(32);
    dim3 grid(1);

    sumArrayOnGpu<<<grid, block>>>(ga, gb, gres);
    hipMemcpy(res, gres, nbyte, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        cout << res[i];
    }

    hipFree(ga);
    hipFree(gb);
    hipFree(gres);
}