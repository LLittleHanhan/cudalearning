#include <hip/hip_runtime.h>

#include <cstring>
#include <iostream>
using namespace std;

void initval(int s[], int n) {
    for (int i = 0; i < n; i++) {
        s[i] = 1;
    }
}

__global__ void sumTwoArrayOnGpu(int* ga, int* gb, int* gres, int tx, int ty, int bx, int by) {
    int i = (blockIdx.y * bx + blockIdx.x) * tx * ty + threadIdx.y * tx + threadIdx.x;
    gres[i] = ga[i] + gb[i];
}

void sumTwoMatrix(int tx, int ty) {
    int l = 1024;
    int n = l * l;
    int nbyte = sizeof(int) * n;

    int* a = new int[n];
    int* b = new int[n];
    int* res = new int[n];
    initval(a, n);
    initval(b, n);

    int *ga, *gb, *gres;
    hipMalloc(&ga, nbyte);
    hipMalloc(&gb, nbyte);
    hipMalloc(&gres, nbyte);

    hipMemcpy(ga, a, nbyte, hipMemcpyHostToDevice);
    hipMemcpy(gb, b, nbyte, hipMemcpyHostToDevice);
    hipMemcpy(gres, a, nbyte, hipMemcpyHostToDevice);

    dim3 block(tx, ty);
    dim3 grid(l / tx, l / ty);
    cout << tx << " " << ty << " " << l / tx << " " << l / ty;
    sumTwoArrayOnGpu<<<grid, block>>>(ga, gb, gres, tx, ty, l / tx, l / ty);
    hipMemcpy(res, gres, nbyte, hipMemcpyDeviceToHost);

    cout << res[n - l] << endl;

    hipFree(ga);
    hipFree(gb);
    hipFree(gres);
    delete[] a;
    delete[] b;
    delete[] res;
}

__global__ void reduce1(int* ga, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int stride = n / 2; stride > 0; stride = (stride + 1) / 2) {
        if (i < stride) {
            ga[i] += ga[i + stride];
        }
        ga[stride] = 0;
        if (stride == 1)
            break;
    }
}

void sumArray(int tx) {
    int n = 1024;
    int nbyte = sizeof(int) * n;
    int* a = new int[n];
    int* res = new int;
    initval(a, n);

    int* ga;
    hipMalloc(&ga, nbyte);
    dim3 block(tx);
    dim3 grid(n / tx);
    cout << tx << " " << n / tx;
    reduce1<<<grid, block>>>(ga, n);
    hipMemcpy(res, ga, sizeof(int), hipMemcpyDeviceToHost);

    cout << *res << endl;

    hipFree(ga);
    delete[] a;
    delete[] res;
}

int main(int argc, char** argv) {
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, dev);
    cout << prop.name << endl;
    cout << "num of mutipro:" << prop.multiProcessorCount << endl;
    cout << "max num threads of per block:" << prop.maxThreadsPerBlock << endl;
    cout << "max num threads of per mutipro:" << prop.maxThreadsPerMultiProcessor << endl;
    cout << "max num warps of per mutipro:" << prop.maxThreadsPerMultiProcessor / 32 << endl;

    int tx = atoi(argv[1]);
    int ty = atoi(argv[2]);
    // sumTwoMatrix(tx, ty);
    sumArray(tx);
}