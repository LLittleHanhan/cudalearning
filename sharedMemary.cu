#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstring>
#include <iostream>
using namespace std;

int main() {
    hipSharedMemConfig conf;
    hipDeviceGetSharedMemConfig(&conf);
    cout << conf;
}