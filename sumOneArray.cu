#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <cstring>
#include <iostream>
using namespace std;

void initval(int s[], int n) {
    for (int i = 0; i < n; i++) {
        s[i] = 1;
    }
}

__global__ void reduce(int* ga, int* gres, int m) {
    int tid = threadIdx.x;
    int* arr = blockIdx.x * blockDim.x * m + ga;
    for (int i = 1; i < m; i++) {
        arr[tid] += arr[tid + blockDim.x * i];
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride)
            arr[tid] += arr[tid + stride];
        __syncthreads();
    }
    if (tid == 0)
        gres[blockIdx.x] = arr[0];
}

void sumOneArray(int tx, int m) {
    int n = 1024 * 1024;
    int* a = new int[n];
    int* res = new int[n / (tx * m)];
    initval(a, n);

    int *ga, *gres;
    hipMalloc(&ga, sizeof(int) * n);
    hipMalloc(&gres, sizeof(int) * n / (tx * m));
    hipMemcpy(ga, a, sizeof(int) * n, hipMemcpyHostToDevice);

    dim3 block(tx);
    dim3 grid(n / (tx * m));
    cout << n / (tx * m) << " " << tx << " " << m << endl;
    reduce2<<<grid, block>>>(ga, gres, m);
    hipMemcpy(res, gres, sizeof(int) * n / (tx * m), hipMemcpyDeviceToHost);

    int sum = 0;
    for (int i = 0; i < n / (tx * m); i++)
        sum += res[i];
    cout << sum << endl;

    hipFree(ga);
    hipFree(gres);
    delete[] a;
    delete[] res;
}

int main(int argc, char** argv) {
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, dev);
    cout << prop.name << endl;
    cout << "num of mutipro:" << prop.multiProcessorCount << endl;
    cout << "max num threads of per block:" << prop.maxThreadsPerBlock << endl;
    cout << "max num threads of per mutipro:" << prop.maxThreadsPerMultiProcessor << endl;
    cout << "max num warps of per mutipro:" << prop.maxThreadsPerMultiProcessor / 32 << endl;

    int tx = atoi(argv[1]);
    int m = atoi(argv[2]);
    sumOneArray(tx, m);
}